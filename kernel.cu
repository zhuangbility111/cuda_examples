#include <hip/hip_runtime.h>

template <int NUM_THREADS_PER_BLOCK>
__global__ void reduce_sum_v0(float *input, float *output, int n) {
    __shared__ float shared_data[NUM_THREADS_PER_BLOCK];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // load data to shared mem
    shared_data[tid] = input[idx];

    // sync
    __syncthreads();

    for (int stride = NUM_THREADS_PER_BLOCK; stride > 0; stride /= 2) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    

}